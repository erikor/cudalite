
#include <hip/hip_runtime.h>
extern "C" 

__global__
void kernexec(double nrow, double ncol, double *x, double *out)
{

   /*
    * striding to allow for matrix larger than number of available threads
    *
    * should produce exact copy of input matrix regardless of size of available device grid
    * inspired by 1D example at: 
    * https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
    */

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    
    for (int i = r; i < nrow; i+= blockDim.x * gridDim.x) {
      for (int j = c; j < ncol; j+= blockDim.y * gridDim.y) {
        int index = i * ncol + j;
        out[index] = x[index];
      }
    }
}
